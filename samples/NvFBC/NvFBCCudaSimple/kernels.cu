#include <assert.h>
#include <hip/hip_runtime.h>


__global__ static void CudaProcess(int w, int h, float * pZBuffer1, float * pZBuffer2, unsigned char * pHostMappedResult)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y; 
    float z1 = pZBuffer1[i+j*w];
    float z2 = pZBuffer2[i+j*w];
    
    // sample processing / image analysis
    if (z1<=z2)		            // pixels are BGRA, so .x is blue component
    {
		// ....
    }
    
    //...
    
    
 }

extern "C" 
{
	hipError_t launch_CudaProcess(int w, int h, float * pZBuffer1, float * pZBuffer2, hipDeviceptr_t pHostMappedResult)
	{
		dim3 dimBlock(16, 16, 1);
		dim3 dimGrid(w/dimBlock.x, h/dimBlock.y, 1);

		CudaProcess<<<dimGrid, dimBlock>>>(w, h, pZBuffer1, pZBuffer2, (unsigned char *)pHostMappedResult);	
		hipError_t err = hipGetLastError();                                
		return err;
	}
}